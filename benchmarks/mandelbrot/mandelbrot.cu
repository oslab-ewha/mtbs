// Note: Most of the code comes from the MacResearch OpenCL podcast
#include <stdio.h>

#include "../benchapi.h"

__device__ int
mandelbrot(void *args[])
{
	char	*out = (char *)args[0];
	int	width = (int)(long long)args[1];
	unsigned int	x_dim = get_blockIdxX() * get_blockDimX() + get_threadIdxX();
	unsigned int	y_dim = get_blockIdxY() * get_blockDimY() + get_threadIdxY();
	int	index = 3 * width * y_dim + x_dim * 3;
	float	x_origin = ((float)x_dim / width) * 3.25 - 2;
	float	y_origin = ((float)y_dim / width) * 2.5 - 1.25;

	float x = 0.0;
	float y = 0.0;

	int	iteration = 0;
	int	max_iteration = 2048;
	while (x * x + y * y <= 14 && iteration < max_iteration) {
		float xtemp = x * x - y * y + x_origin;
		y = 2 * x * y + y_origin;
		x = xtemp;
		iteration++;
	}

	if (iteration == max_iteration) {
		out[index] = 0;
		out[index + 1] = 0;
		out[index + 2] = 0;
	} else {
		out[index] = iteration;
		out[index + 1] = iteration;
		out[index + 2] = iteration;
	}
	return iteration;
}

int
bench_mandelbrot(dim3 dimGrid, dim3 dimBlock, void *args[])
{
	int	width;
	int	height;
	char	*image, *host_image;
	size_t	buffer_size;
	int		res;
	skrid_t		skrid;
	vstream_t	strm;

	width = dimGrid.x * dimBlock.x;
	height = dimGrid.y * dimBlock.y;

	// Multiply by 3 here, since we need red, green and blue for each pixel
	buffer_size = sizeof(char) * width * height * 3;

	if (hipMalloc((void **)&image, buffer_size) != hipSuccess) {
		printf("failed to allocate mem: buffer size: %ld\n", buffer_size);
		return -1;
	}
	host_image = (char *)malloc(buffer_size);

	args[0] = image;
	args[1] = (void *)(long long)width;
	args[2] = (void *)(long long)height;

	strm = create_vstream();
	skrid = launch_kernel(MANDELBROT, strm, dimGrid, dimBlock, args);
	wait_kernel(skrid, strm, &res);

	hipMemcpyAsync(host_image, image, buffer_size, hipMemcpyDeviceToHost, *(hipStream_t *)strm);
	hipStreamSynchronize(*(hipStream_t *)strm);

	destroy_vstream(strm);

#if 0
	//TODO: hipFree freeze the system
	hipFree(image);
#endif
	free(host_image);

	return res;
}
