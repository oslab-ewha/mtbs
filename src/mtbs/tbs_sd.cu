#include "hip/hip_runtime.h"
#include "mtbs_cu.h"

extern void init_sched(void);
extern void fini_sched(void);

extern void wait_fedkern_initialized(fedkern_info_t *d_fkinfo);

extern __device__ void try_setup_dyn_sched(fedkern_info_t *fkinfo);

extern __device__ skrid_t get_skrid_dyn(void);
extern __device__ void advance_epoch_dyn(skrid_t skrid);

__device__ BOOL	going_to_shutdown;

extern "C" __global__ void
func_macro_TB(fedkern_info_t *fkinfo)
{
	if (threadIdx.x == 0 && threadIdx.y == 0) {
		try_setup_dyn_sched(fkinfo);
	}
	__syncthreads();

	while (!going_to_shutdown) {
		skrid_t	skrid;
		skrun_t	*skr;

		skrid = get_skrid_dyn();
		if (skrid == 0)
			return;

		skr = &d_skruns[skrid - 1];
		run_sub_kernel(skr);

		advance_epoch_dyn(skrid);
	}
}

static BOOL
launch_macro_TB(fedkern_info_t *fkinfo)
{
	hipStream_t	strm;
	hipError_t	err;
	hipFunction_t	func_macro_TB;
	void	*params[1];

	hipStreamCreateWithFlags(&strm, hipStreamNonBlocking);
	hipModuleGetFunction(&func_macro_TB, mod, "func_macro_TB");

	params[0] = &fkinfo;
	err = hipModuleLaunchKernel(func_macro_TB, n_sm_count, n_MTBs_per_sm, 1,
			     n_threads_per_MTB, 1, 1, 0, strm, params, NULL);
	if (err != hipSuccess) {
		error("kernel launch error: %s\n", get_cuda_error_msg(err));
		return FALSE;
	}

	wait_fedkern_initialized(fkinfo);
	return TRUE;
}

static void
stop_macro_TB(fedkern_info_t *fkinfo)
{
	BOOL	done = TRUE;

	hipMemcpyHtoD((hipDeviceptr_t)&fkinfo->sched_done, &done, sizeof(BOOL));
}

BOOL
run_sd_tbs(unsigned *pticks)
{
	fedkern_info_t	*fkinfo;

	init_sched();

	fkinfo = create_fedkern_info();

	if (!launch_macro_TB(fkinfo))
		return FALSE;

	start_benchruns();

	init_tickcount();

	wait_benchruns();

	*pticks = get_tickcount();

	fini_sched();

	stop_macro_TB(fkinfo);

	free_fedkern_info(fkinfo);

	return TRUE;
}
