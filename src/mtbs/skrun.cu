#include "hip/hip_runtime.h"
#include "mtbs_cu.h"

#include <pthread.h>
#include <unistd.h>

#include <hip/hip_runtime.h>

#include "stream.h"

hipCtx_t	context;

__device__ tbs_type_t	d_tbs_type;
__device__ skrun_t	*d_skruns;
__device__ unsigned	*d_mtbs_done_cnts;

static skrun_t	*g_skruns;
static unsigned	*g_mtbs_done_cnts;

static unsigned	*info_n_mtbs;

static BOOL	*skrun_dones;
static unsigned	skrid_done_min;
static unsigned	cur_skrid_host;

static BOOL	checker_done;
static pthread_t	checker;
static pthread_mutex_t	mutex = PTHREAD_MUTEX_INITIALIZER;
static pthread_cond_t	cond = PTHREAD_COND_INITIALIZER;

static hipStream_t	strm_submit;

#define SK_PROTO(name)	__device__ int name(void *args[])
#define SK_FUNCS(base)	SK_PROTO(base);

SK_FUNCS(loopcalc)
SK_FUNCS(mklc)
SK_FUNCS(gma)
SK_FUNCS(lma)
SK_FUNCS(kmeans)
SK_FUNCS(mandelbrot)

static __device__ int
run_sub_kernel_func(skid_t skid, void *args[])
{
	switch (skid) {
	case LOOPCALC:
		return loopcalc(args);
	case MKLC:
		return mklc(args);		
	case GMA:
		return gma(args);
	case LMA:
		return lma(args);
	case KMEANS:
		return kmeans(args);
	case MANDELBROT:
		return mandelbrot(args);
	default:
		return 0;
	}
}

__device__ void
run_sub_kernel(skrid_t skrid)
{
	skrun_t	*skr;
	int	res;

	skr = &d_skruns[skrid - 1];
	res = run_sub_kernel_func(skr->skid, (void **)skr->args);
	if (get_blockIdxX() == 0 && get_blockIdxY() == 0 && get_threadIdxX() == 0 && get_threadIdxY() == 0) {
		skr->res = res;
	}
}

__global__ void
sub_kernel_func(skrid_t skrid)
{
	run_sub_kernel(skrid);
}

static skrid_t
submit_skrun(skid_t skid, dim3 dimGrid, dim3 dimBlock, void *args[])
{
	skrid_t	skrid;
	skrun_t	skrun;

	skrun.skid = skid;
	skrun.dimGrid = dimGrid;
	skrun.dimBlock = dimBlock;
	memcpy(skrun.args, args, sizeof(void *) * MAX_ARGS);
	skrun.res = 0;
	skrun.n_tbs = dimGrid.x * dimGrid.y;
	skrun.n_mtbs_per_tb = dimBlock.x * dimBlock.y / N_THREADS_PER_mTB;

	pthread_mutex_lock(&mutex);

	skrid = cur_skrid_host + 1;
	info_n_mtbs[skrid - 1] = skrun.n_tbs * skrun.n_mtbs_per_tb;

	hipMemcpyAsync(g_skruns + cur_skrid_host, &skrun, sizeof(skrun_t), hipMemcpyHostToDevice, strm_submit);
	hipStreamSynchronize(strm_submit);

	cur_skrid_host++;

	if (sched->type == TBS_TYPE_SD_STATIC) {
		extern void schedule_mtbs(skrid_t skrid, unsigned n_tbs, unsigned n_mtbs_per_tb);
		schedule_mtbs(skrid, skrun.n_tbs, skrun.n_mtbs_per_tb);
	}

	pthread_mutex_unlock(&mutex);

	return skrid;
}

skrid_t
launch_kernel(skid_t skid, vstream_t strm, dim3 dimGrid, dim3 dimBlock, void *args[])
{
	skrid_t	skrid;
	hipStream_t	cstrm = NULL;

	skrid = submit_skrun(skid, dimGrid, dimBlock, args);

	if (sched->type == TBS_TYPE_HW) {
		if (strm != NULL) {
			cstrm = ((vstrm_t)strm)->cudaStrm;
		}
		sub_kernel_func<<<dimGrid, dimBlock, 0, cstrm>>>(skrid);
	}
	return skrid;
}

static void
wait_skrun(skrid_t skrid)
{
	pthread_mutex_lock(&mutex);

	while (!checker_done && !skrun_dones[skrid - 1])
		pthread_cond_wait(&cond, &mutex);

	pthread_mutex_unlock(&mutex);
}

void
wait_kernel(skrid_t skrid, vstream_t strm, int *pres)
{
	skrun_t	*skr;
	int	res;

	if (sched->type == TBS_TYPE_HW) {
		if (strm != NULL) {
			hipStreamSynchronize(((vstrm_t)strm)->cudaStrm);
		}
		else {
			hipDeviceSynchronize();
		}
	}
	else
		wait_skrun(skrid);

	skr = g_skruns + (skrid - 1);
	hipMemcpyAsync(&res, &skr->res, sizeof(int), hipMemcpyDeviceToHost, strm_submit);
	hipStreamSynchronize(strm_submit);
	*pres = res;
}

static void
notify_done_skruns(unsigned *mtbs_done_cnts, unsigned n_checks)
{
	unsigned	min_new = skrid_done_min;
	BOOL		notify = FALSE;
	unsigned	i;

	pthread_mutex_lock(&mutex);

	for (i = 0; i < n_checks; i++) {
		if (skrun_dones[i + skrid_done_min])
			continue;
		if (mtbs_done_cnts[i] == info_n_mtbs[i + skrid_done_min]) {
			notify = TRUE;
			skrun_dones[i + skrid_done_min] = TRUE;
			if (min_new == i + skrid_done_min) {
				min_new++;
			}
		}
	}
	skrid_done_min = min_new;
	if (notify)
		pthread_cond_broadcast(&cond);
	pthread_mutex_unlock(&mutex);
}

static void *
skruns_checkfunc(void *arg)
{
	hipStream_t	strm;

	hipCtxSetCurrent(context);

	hipStreamCreate(&strm);

	while (!checker_done) {
		unsigned	n_checks = cur_skrid_host - skrid_done_min;
		if (n_checks > 0) {
			notify_done_skruns(g_mtbs_done_cnts, n_checks);
		}
		usleep(100);
	}

	hipStreamDestroy(strm);
	return NULL;
}

__global__ void
kernel_init_skrun(tbs_type_t type, skrun_t *skruns, unsigned *mtbs_done_cnts)
{
	int	i;

	d_tbs_type = type;
	d_skruns = skruns;
	d_mtbs_done_cnts = mtbs_done_cnts;
	for (i = 0; i < MAX_QUEUED_KERNELS; i++) {
		skruns[i].skid = 0;
		mtbs_done_cnts[i] = 0;
	}
}

void
init_skrun(void)
{
	hipError_t	err;

	hipCtxGetCurrent(&context);

	hipStreamCreate(&strm_submit);

	hipMalloc(&g_skruns, sizeof(skrun_t) * MAX_QUEUED_KERNELS);
	hipHostMalloc(&g_mtbs_done_cnts, sizeof(unsigned) * MAX_QUEUED_KERNELS);

	info_n_mtbs = (unsigned *)calloc(MAX_QUEUED_KERNELS, sizeof(unsigned));
	skrun_dones = (BOOL *)calloc(MAX_QUEUED_KERNELS, sizeof(BOOL));

	pthread_create(&checker, NULL, skruns_checkfunc, NULL);

	dim3 dimGrid(1,1), dimBlock(1,1);
	kernel_init_skrun<<<dimGrid, dimBlock>>>(sched->type, g_skruns, g_mtbs_done_cnts);
	err = hipGetLastError();
	if (err != hipSuccess)
		error("failed to initialize skrun: %s\n", hipGetErrorString(err));
	else
		hipDeviceSynchronize();
}

void
fini_skrun(void)
{
	void	*retval;

	checker_done = TRUE;
	pthread_join(checker, &retval);
}
