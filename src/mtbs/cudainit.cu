#include "hip/hip_runtime.h"
#include "mtbs_cu.h"

hipModule_t	mod;

extern "C" unsigned	arg_n_MTBs_per_sm;
extern "C" unsigned	arg_n_threads_per_MTB;

static BOOL
setup_gpu_devinfo(hipDevice_t dev)
{
	unsigned	max_threads_per_sm, max_threads_per_block;
	hipError_t	err;

	err = hipDeviceGetAttribute((int *)&n_sm_count, hipDeviceAttributeMultiprocessorCount, dev);
	if (err != hipSuccess) {
		error("failed to get gpu device property(sm count): %s", get_cuda_error_msg(err));
		return FALSE;
	}

	err = hipDeviceGetAttribute((int *)&max_threads_per_block, hipDeviceAttributeMaxThreadsPerBlock, dev);
	if (err != hipSuccess) {
		error("failed to get gpu device property(max threads per block): %s", get_cuda_error_msg(err));
		return FALSE;
	}
	err = hipDeviceGetAttribute((int *)&max_threads_per_sm, hipDeviceAttributeMaxThreadsPerMultiProcessor, dev);
	if (err != hipSuccess) {
		error("failed to get gpu device property(max threads per sm): %s", get_cuda_error_msg(err));
		return FALSE;
	}

	if (arg_n_MTBs_per_sm == 0 && arg_n_threads_per_MTB == 0) {
		n_threads_per_MTB = max_threads_per_block;
		n_MTBs_per_sm = max_threads_per_sm / n_threads_per_MTB;
	}
	else if (arg_n_MTBs_per_sm > 0) {
		n_MTBs_per_sm = arg_n_MTBs_per_sm;
		if (arg_n_threads_per_MTB > 0)
			n_threads_per_MTB = arg_n_threads_per_MTB;
		else
			n_threads_per_MTB = max_threads_per_sm / n_MTBs_per_sm;
	}
	else {
		n_threads_per_MTB = arg_n_threads_per_MTB;
		n_MTBs_per_sm = max_threads_per_sm / n_threads_per_MTB;
	}

	if (n_threads_per_MTB > max_threads_per_block)
		n_threads_per_MTB = max_threads_per_block;
	if (n_threads_per_MTB < 32) {
		error("Too small threads per MTB: %d", n_threads_per_MTB);
		return FALSE;
	}
	if (n_threads_per_MTB % 32) {
		error("Invalid thread count per MTB: %d", n_threads_per_MTB);
		return FALSE;
	}

	return TRUE;
}

BOOL
init_cuda(void)
{
	hipError_t	res;
	hipDevice_t	dev;

	hipInit(0);
	res = hipDeviceGet(&dev, devno);
	if (res != hipSuccess) {
		error("failed to get device: %s", get_cuda_error_msg(res));
		return FALSE;
	}

	if (!setup_gpu_devinfo(dev)) {
		return FALSE;
	}

	res = hipDevicePrimaryCtxRetain(&context, dev);
	if (res != hipSuccess) {
		error("failed to get context: %s", get_cuda_error_msg(res));
		return FALSE;
	}

	res = hipCtxSetCurrent(context);
	if (res != hipSuccess) {
		error("failed to set context: %s", get_cuda_error_msg(res));
		return FALSE;
	}

	res = hipModuleLoad(&mod, "mtbs.cubin");
	if (res != hipSuccess) {
		error("failed to load module mtbs.cubin: %s", get_cuda_error_msg(res));
		return FALSE;
	}

	return TRUE;
}
