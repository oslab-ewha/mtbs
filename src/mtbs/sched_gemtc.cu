#include "mtbs_cu.h"

#include <pthread.h>

#include "sched_gemtc.cuh"

static skrun_t	*g_skruns;
static BOOL	*g_mtbs_done;

static BOOL	*skrun_dones;

static pthread_mutex_t	mutex = PTHREAD_MUTEX_INITIALIZER;
static pthread_cond_t	cond = PTHREAD_COND_INITIALIZER;

static unsigned	skrid_done_min;
static unsigned	cur_skrid_host;

static BOOL	checker_done;
static pthread_t	checker;

static hipStream_t	strm_gemtc;

static sk_t
submit_skrun_gemtc(vstream_t vstream, skrun_t *skr)
{
	skrid_t	skrid;

	pthread_mutex_lock(&mutex);

	while (skrid_done_min == (cur_skrid_host + 1) % n_queued_kernels) {
		/* full */
		pthread_cond_wait(&cond, &mutex);
	}

	skrid = cur_skrid_host + 1;
	skrun_dones[skrid - 1] = FALSE;
	hipMemcpyHtoDAsync((hipDeviceptr_t)(g_skruns + cur_skrid_host), skr, sizeof(skrun_t), strm_gemtc);
	/* No synchronization needed */

	cur_skrid_host = (cur_skrid_host + 1) % n_queued_kernels;

	pthread_mutex_unlock(&mutex);

	return (sk_t)(long long)skrid;
}

static void
wait_skrun_gemtc(sk_t sk, vstream_t vstream, int *pres)
{
	skrun_t	*skr;
	skrid_t	skrid = (skrid_t)(long long)sk;

	pthread_mutex_lock(&mutex);

	while (!checker_done && !skrun_dones[skrid - 1])
		pthread_cond_wait(&cond, &mutex);

	pthread_mutex_unlock(&mutex);

	skr = g_skruns + (skrid - 1);
	hipMemcpyDtoHAsync(pres, (hipDeviceptr_t)&skr->res, sizeof(int), strm_gemtc);
	hipStreamSynchronize(strm_gemtc);
}

static void
notify_done_skruns(unsigned n_checks)
{
	unsigned	min_new = skrid_done_min;
	BOOL		notify = FALSE;
	unsigned	i, idx;

	idx = skrid_done_min;
	for (i = 0; i < n_checks; i++) {
		if (!skrun_dones[idx]) {
			if (g_mtbs_done[idx]) {
				notify = TRUE;
				skrun_dones[idx] = TRUE;
				g_mtbs_done[idx] = FALSE;
			}
		}
		if (skrun_dones[idx]) {
			if (min_new == idx) {
				min_new = (min_new + 1) % n_queued_kernels;
				notify = TRUE;
			}
		}
		idx = (idx + 1) % n_queued_kernels;
	}
	skrid_done_min = min_new;
	if (notify)
		pthread_cond_broadcast(&cond);
}

static void *
skruns_checkfunc(void *arg)
{
	while (!checker_done) {
		unsigned	n_checks = (cur_skrid_host + n_queued_kernels - skrid_done_min) % n_queued_kernels;
		pthread_mutex_lock(&mutex);

		if (n_checks > 0) {
			notify_done_skruns(n_checks);
		}

		pthread_mutex_unlock(&mutex);
		usleep(100);
	}

	return NULL;
}

static void
init_skrun_gemtc(void)
{
	void	*params[2];
	unsigned	i;

	hipStreamCreateWithFlags(&strm_gemtc, hipStreamNonBlocking);

	g_skruns = (skrun_t *)mtbs_cudaMalloc(sizeof(skrun_t) * n_queued_kernels);
	hipMemAllocHost((void **)&g_mtbs_done, sizeof(BOOL) * n_queued_kernels);
	for (i = 0; i < n_queued_kernels; i++) {
		g_mtbs_done[i] = FALSE;
	}

	skrun_dones = (BOOL *)calloc(n_queued_kernels, sizeof(BOOL));

	pthread_create(&checker, NULL, skruns_checkfunc, NULL);

	params[0] = &g_skruns;
	params[1] = &g_mtbs_done;
	if (!invoke_kernel_func("setup_sched_gemtc", params)) {
		exit(12);
	}
}

static void
fini_skrun_gemtc(void)
{
	void	*retval;

	checker_done = TRUE;
	pthread_join(checker, &retval);
	mtbs_cudaFree(g_skruns);
}

sched_t	sched_sd_gemtc = {
	"gemtc",
	TBS_TYPE_SD_GEMTC,
	"func_macro_TB_gemtc",
	init_skrun_gemtc,
	fini_skrun_gemtc,
	submit_skrun_gemtc,
	wait_skrun_gemtc,
};
